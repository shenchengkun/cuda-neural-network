#include <dataset.cuh>
#include <mnist.cuh>

#define BATCH_SIZE 128
#define LEARNING_RATE 0.003
#define L2 0.0001
#define EPOCHS 30
#define BETA 0.99

// test
int main() {
  // DataSet dataset("./mnist_data", true);
  // dataset.forward(64, true);
  // dataset.print_im();

  auto cudaStatus = hipSetDevice(0);
  CHECK_EQ(cudaStatus, hipSuccess,
           "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");

  Minist mnist("./mnist_data", LEARNING_RATE, L2, BETA);
  mnist.train(EPOCHS, BATCH_SIZE);
}
